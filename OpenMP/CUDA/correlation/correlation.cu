#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define POLYBENCH_TIME 1



/* Include polybench common header. */
#include <polybench.h>
#include <polybench.c>
#define POLYBENCH_TIME 1

typedef double DATA_TYPE; // Definizione di DATA_TYPE

/* Include benchmark-specific header. */
/* Default data type is double, default size is 1000. */
//#define MINI_DATASET
//#define SMALL_DATASET
//#define STANDARD_DATASET
//#define LARGE_DATASET
//#define EXTRALARGE_DATASET

#include "correlation.cuh"

/* Dimensioni predefinite */
#define BLOCK_SIZE 32  // Dimensione blocchi

__global__ void compute_mean(int m, int n, DATA_TYPE* data, DATA_TYPE* mean, DATA_TYPE float_n) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < m) {
        DATA_TYPE sum = 0.0;
        for (int i = 0; i < n; i++) {
            sum += data[i * m + j];
        }
        mean[j] = sum / float_n;
    }
}

__global__ void compute_stddev(int m, int n, DATA_TYPE* data, DATA_TYPE* mean, DATA_TYPE* stddev, DATA_TYPE float_n, DATA_TYPE eps) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < m) {
        DATA_TYPE sum = 0.0;
        for (int i = 0; i < n; i++) {
            DATA_TYPE diff = data[i * m + j] - mean[j];
            sum += diff * diff;
        }
        stddev[j] = sqrt(sum / float_n);
        stddev[j] = (stddev[j] <= eps) ? 1.0 : stddev[j];
    }
}

__global__ void normalize_data(int m, int n, DATA_TYPE* data, DATA_TYPE* mean, DATA_TYPE* stddev, DATA_TYPE float_n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < m) {
        data[i * m + j] = (data[i * m + j] - mean[j]) / (sqrt(float_n) * stddev[j]);
    }
}

__global__ void compute_correlation(int m, int n, DATA_TYPE* data, DATA_TYPE* symmat) {
    int j1 = blockIdx.x * blockDim.x + threadIdx.x;
    int j2 = blockIdx.y * blockDim.y + threadIdx.y;

    if (j1 < m && j2 < m && j1 <= j2) {
        DATA_TYPE sum = 0.0;
        for (int i = 0; i < n; i++) {
            sum += data[i * m + j1] * data[i * m + j2];
        }
        symmat[j1 * m + j2] = sum;
        symmat[j2 * m + j1] = sum;  // Simmetrico
    }
}

/* Inizializzazione dell'array */
static void init_array(int m, int n, DATA_TYPE *float_n, DATA_TYPE POLYBENCH_2D(data, M, N, m, n)) {
    *float_n = 1.2;
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
            data[i][j] = ((DATA_TYPE)i * j) / M;
}

/* Stampa della matrice */
static void print_array(int m, DATA_TYPE POLYBENCH_2D(symmat, M, M, m, m)) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < m; j++)
            printf("%f ", symmat[i][j]);
        printf("\n");
    }
}

int main(int argc, char** argv) {
    /* Recupera le dimensioni del problema. */
    int n = N;
    int m = M;

    /* Dichiarazione/allocazione variabili. */
    DATA_TYPE float_n;
    POLYBENCH_2D_ARRAY_DECL(data, DATA_TYPE, M, N, m, n);
    POLYBENCH_2D_ARRAY_DECL(symmat, DATA_TYPE, M, M, m, m);
    POLYBENCH_1D_ARRAY_DECL(mean, DATA_TYPE, M, m);
    POLYBENCH_1D_ARRAY_DECL(stddev, DATA_TYPE, M, m);

    /* Inizializzazione degli array. */
    init_array(m, n, &float_n, POLYBENCH_ARRAY(data));

    /* Allocazione GPU */
    DATA_TYPE *d_data, *d_symmat, *d_mean, *d_stddev;
    hipMalloc((void**)&d_data, m * n * sizeof(DATA_TYPE));
    hipMalloc((void**)&d_symmat, m * m * sizeof(DATA_TYPE));
    hipMalloc((void**)&d_mean, m * sizeof(DATA_TYPE));
    hipMalloc((void**)&d_stddev, m * sizeof(DATA_TYPE));

    /* Copia dati su GPU */
    hipMemcpy(d_data, POLYBENCH_ARRAY(data), m * n * sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    /* Configurazione dei kernel */
    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim1D((m + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 gridDim2D((m + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);

    /* Start timer. */
    polybench_start_instruments;

    /* Lancio kernel per media */
    compute_mean<<<gridDim1D, blockDim>>>(m, n, d_data, d_mean, float_n);
    hipDeviceSynchronize();

    /* Lancio kernel per deviazione standard */
    compute_stddev<<<gridDim1D, blockDim>>>(m, n, d_data, d_mean, d_stddev, float_n, 0.1);
    hipDeviceSynchronize();

    /* Lancio kernel per normalizzazione */
    normalize_data<<<gridDim2D, blockDim>>>(m, n, d_data, d_mean, d_stddev, float_n);
    hipDeviceSynchronize();

    /* Lancio kernel per calcolo correlazione */
    compute_correlation<<<gridDim2D, blockDim>>>(m, n, d_data, d_symmat);
    hipDeviceSynchronize();

    /* Copia risultati su CPU */
    hipMemcpy(POLYBENCH_ARRAY(symmat), d_symmat, m * m * sizeof(DATA_TYPE), hipMemcpyDeviceToHost);

    /* Stop and print timer. */
    polybench_stop_instruments;
    polybench_print_instruments;

    /* Prevent dead-code elimination. All live-out data must be printed
    by the function call in argument. */
    polybench_prevent_dce(print_array(m, POLYBENCH_ARRAY(symmat)));
    //print_array(m, POLYBENCH_ARRAY(symmat));

    /* Pulizia memoria */
    POLYBENCH_FREE_ARRAY(data);
    POLYBENCH_FREE_ARRAY(symmat);
    POLYBENCH_FREE_ARRAY(mean);
    POLYBENCH_FREE_ARRAY(stddev);
    hipFree(d_data);
    hipFree(d_symmat);
    hipFree(d_mean);
    hipFree(d_stddev);

    return 0;
}